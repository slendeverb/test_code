#include "header.h"

int main() {
    thrust::universal_vector<float> in;
    thrust::universal_vector<float> out;
    auto [nx,ny,comp]=read_image(in,"../../sources/img/original.jpg");
    out.resize(in.size());

    TICK(parallel_jacobi);
    constexpr int iters=4;
    for (int step=0;step<256;step+=iters) {
        parallel_jacobi<iters,32>(out.data().get(),in.data().get(),nx,ny);
        thrust::swap(out,in);
    }
    checkCudaErrors(hipDeviceSynchronize());
    TOCK(parallel_jacobi);

    write_image(in,nx,ny,1,"../../sources/img/out.png");
    cv::Mat image=cv::imread("../../sources/img/out.png");
    int new_width=600;
    int new_height=image.rows/((float)image.cols/new_width);
    cv::Mat resized_image;
    cv::resize(image,resized_image,cv::Size(new_width,new_height));
    cv::imshow("resized_image",resized_image);
    cv::waitKey(0);
}