#include "hip/hip_runtime.h"
#include "header.h"

__host__ __device__ void say_hello() {
#ifdef __CUDA_ARCH__
    printf("Hello World from GPU architecture %d!\n",__CUDA_ARCH__);
#else
    printf("Hello World from CPU!\n");
#endif
}

__global__ void kernel() {
    say_hello();
}

int main() {
    kernel<<<1,1>>>();
    hipDeviceSynchronize();
    say_hello();
    std::unordered_set<int> ust;
    auto x=ust.insert(1).first;
    SHOW(decltype(x));
}